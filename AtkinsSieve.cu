
#include <hip/hip_runtime.h>


bool* sieveOfAtkin(const int n) {
    bool* numbers = new bool[n];


    for (int i = 0; i <= n; ++i) {
        numbers[i] = false;
    }

    int sqrtN = sqrt(n);

    for (int x = 1; x <= sqrtN; x++) {
        for (int y = 1; y <= sqrtN; y++) {
            int z = 4 * x * x + y * y;
            if (z <= n && (z % 12 == 1 || z % 12 == 5)) {
                numbers[z] = !numbers[z];
            }
            z = 3 * x * x + y * y;
            if (z <= n && z % 12 == 7) {
                numbers[z] = !numbers[z];
            }
            z = 3 * x * x - y * y;
            if (x > y && z <= n && z % 12 == 11) {
                numbers[z] = !numbers[z];
            }
        }
    }

    numbers[3] = numbers[2] = true;

    for (int x = 2; x <= sqrtN; x++) {
        if (numbers[x]) {
            // Corrected loop to mark multiples of primes
            for (int y = x * x; y <= n; y += x) {
                numbers[y] = false;
            }
        }
    }

    return numbers;
}