#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <cmath>

#include <chrono>
#include <stdio.h>

#include "SingleSieves.cu"


__global__ void residue_sieve_ker_1(bool* prime, int x, int n) {
    int y = threadIdx.x;
    if (y * y <= n) {
        int z = 4 * x * x + y * y;
        if ((z <= n) && ((z % 12 == 1) || (z % 12 == 5))) {
            prime[z] = !prime[z];
        }
    }
}

__global__ void residue_sieve_ker_2(bool* prime, int x, int n) {
    int y = threadIdx.x;
    if (y * y <= n) {
        int z = 3 * x * x + y * y;
        if ((z <= n) && (z % 12 == 7)) {
            prime[z] = !prime[z];
        }
    }
}

__global__ void residue_sieve_ker_3(bool* prime, int x, int n) {
    int y = threadIdx.x;
    if (y * y <= n) {
        int z = 3 * x * x - y * y;
        if ((z <= n) && ((z % 12 == 11) && (x > y))) {
            prime[z] = !prime[z];
        }
    }
}



__global__ void remove_squares(bool* prime, int x, int n) {
    int y = threadIdx.x;
    if ((y * y < n) && ((y != 0) || (y != 1))) {
        int x_squared = x * x;
        int remove_square = x_squared * y;
        prime[remove_square] = false;
    }
}

__global__ void initialize(bool* prime, int n, bool value, int y) {
    int x = threadIdx.x;
    if (x < n) {
        prime[y * x] = value;
    }
}

bool* multithreaded_atkins_sieve(const int n = 100, int blockSize = 256) {
    size_t size = n * sizeof(bool);
    int threads = static_cast<int>(std::sqrt(n));

    bool* primes = (bool*)malloc(size);

    bool* d_primes;
    hipMalloc((void**)&d_primes, size);
    hipMemcpy(d_primes, primes, size, hipMemcpyHostToDevice);


    int numBlock = (n + blockSize - 1) / blockSize;

    for (int i = 0; i <= threads; i++) {
        initialize << < 1, threads >> > (d_primes, threads, false, i);
    }

    // this uses n threads but can be optimised for sqrtN threads

  //hipMemcpy(primes, d_primes, size, hipMemcpyDeviceToHost);

  /*
  for (int i = 0; i < n; i++) {
      if (primes[i]) {
          std::cout << "error";
      }
  }
  */


    numBlock = (threads + blockSize - 1) / blockSize;

    //First step in Atkins, remove residues
    for (int x = 0; x * x <= n; x++) {

        //std::cout << "x = " << x << "\n" << "n = " << n << "\n" << "threads = " << threads << "\nnumBlock = " << numBlock << "\n\n";

        residue_sieve_ker_1 << <1, threads >> > (d_primes, x, n);
        residue_sieve_ker_2 << <1, threads >> > (d_primes, x, n);
        residue_sieve_ker_3 << <1, threads >> > (d_primes, x, n);
    }

    //hipMemcpy(primes, d_primes, size, hipMemcpyDeviceToHost);

    //Print primes without removing squares
    /*
    printf("Prime numbers with prime squares on the GPU:\n");
    for (int i = 0; i < n; ++i) {
        if (primes[i]) {
            std::cout << i << "\n";
        }
    }
    */


    //Second step in Atkins, Remove primes
    for (int x = 2; x * x < n; x++) {

        int threads = static_cast<int>(std::sqrt(n));
        int numBlock = (threads + blockSize - 1) / blockSize;

        if (primes[x]) {
            remove_squares << <1, threads >> > (d_primes, x, n);
        }
    }


    hipMemcpy(primes, d_primes, size, hipMemcpyDeviceToHost);

    /*
    printf("Prime numbers on the GPU:\n");
    for (int i = 0; i < n; ++i) {
        if (primes[i]) {
            std::cout << i << "\n";
        }
    }

    */
    hipFree(d_primes);

    primes[3] = primes[2] = true;
    primes[1] = false;


    return primes;
}


__global__ void divisibility_ker(bool* prime, int i, int n) {
    int j = threadIdx.x;
    if ((j > 1) && (j * i < n)) {
        prime[i * j] = false;
    }
}

bool* multithreaded_divisibility_testing(const int n = 100, int blockSize = 256) {

    size_t size = n * sizeof(bool);
    bool* primes = (bool*)malloc(size);

    bool* d_primes;
    hipMalloc((void**)&d_primes, size);
    hipMemcpy(d_primes, primes, size, hipMemcpyHostToDevice);


    int numBlock = (n + blockSize - 1) / blockSize;

    initialize << < 1, n >> > (d_primes, n, true, 1); // this uses n threads but can be optimised for sqrtN threads 

    int threads = n / 2;

    //numBlock = (threads + blockSize - 1) / blockSize;

    int sqrtN = sqrt(n);


    for (int i = 2; i < sqrtN; i++) {

        divisibility_ker << < 1, sqrtN >> > (d_primes, i, n);

    }

    hipMemcpy(primes, d_primes, size, hipMemcpyDeviceToHost);

    hipFree(d_primes);

    return primes;
}




int print(bool* numbers, int n) {
    for (int i = 2; i < n; i++) {
        if (numbers[i]) {
            std::cout << i << "\n";
        }
    }

    return 0;
}

double* time_it(int* cases, int num_cases) {
    double* times = new double[num_cases];


    for (int i = 0; i < num_cases; i++) {
        auto start = std::chrono::high_resolution_clock::now();

        multithreaded_atkins_sieve(cases[i]);

        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
        times[i] = duration.count();
    }

    return times;
}

int main() {
    int n = 30;
    int block_size = 256;
    std::cout << sieveOfAtkin(10);



    int cases[] = { pow(2,20), pow(2,21), pow(2,22), pow(2,23), pow(2,24), pow(2,25), pow(2,26), pow(2,27), pow(2,28), pow(2,29)
        , pow(2,30), /*pow(2,31), pow(2,32), pow(2,34), pow(2,36), pow(2,38), pow(2,40)*/ };
    int num_cases = sizeof(cases) / sizeof(cases[0]);
    double* times = time_it(cases, num_cases);

    std::cout << "cases = [";
    for (int i = 0; i < num_cases; i++) {
        std::cout << " " << cases[i] << ",";
    }
    std::cout << "] \n";

    std::cout << "times = [";
    for (int i = 0; i < num_cases; i++) {
        std::cout << " " << times[i] << ",";
    }
    std::cout << "] \n";




    std::cout << "fin";



    return 0;

}