#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void initialize(bool* prime, int n, bool value, int y) {
    int x = threadIdx.x;
    if (x < n) {
        prime[y * x] = value;
    }
}